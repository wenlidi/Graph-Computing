// *****************************************************************************
// Filename:    message_content_manager.cc
// Date:        2013-01-08 10:06
// Author:      Guangda Lai
// Email:       lambda2fei@gmail.com
// Description: TODO(laigd): Put the file description here.
// *****************************************************************************

#include "message_content_manager.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/gather.h>

#include "constants.h"
#include "device_graph_data_types.h"
#include "device_util.h"

#ifdef LAMBDA_DEBUG
#include "debug.h"
#define LAMBDA_HEADER "------> "
#endif

#define COPY_FROM_DEVICE_TO_DEVICE( \
    FROM, TO, MEMBER, FROM_OFFSET, TO_OFFSET, COUNT, TYPE) \
    checkCudaErrors(hipMemcpyAsync( \
            TO->MEMBER + TO_OFFSET, \
            FROM.MEMBER + FROM_OFFSET, \
            COUNT * sizeof(TYPE), \
            hipMemcpyDeviceToDevice))


unsigned int RoundUpToMultiples(
    const size_t type_size,
    const unsigned int count,
    const size_t bench_mark) {
  return (type_size * count + bench_mark - 1) / bench_mark;
}


void MessageContentManager::Allocate(
    const unsigned int size,
    MessageContent *mcon) {
  mcon->d_size = size;

#ifdef LAMBDA_SHARE_ONE_MESSAGE_ARRAY

  mcon->d_space_size = 0
#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
     + RoundUpToMultiples(sizeof(bool), size, sizeof(unsigned int))
#endif

     //// TODO(laigd): add user defined members
$$M[[+ RoundUpToMultiples(sizeof(<GP_TYPE>), size, sizeof(unsigned int))]]
     ;

  ALLOCATE_ON_DEVICE(unsigned int, mcon->d_space, mcon->d_space_size);

  unsigned int offset = 0;

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  mcon->d_is_full = (bool*)(mcon->d_space + offset);
  offset += RoundUpToMultiples(sizeof(bool), size, sizeof(unsigned int));
#endif

  //// TODO(laigd): add user defined members
$$M[[mcon->d_<GP_NAME> = (<GP_TYPE>*)(mcon->d_space + offset); offset += RoundUpToMultiples(sizeof(<GP_TYPE>), size, sizeof(unsigned int));]]

#else  // Not share one array

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  ALLOCATE_ON_DEVICE(bool,         mcon->d_is_full, mcon->d_size);
#endif

  //// TODO(laigd): add user defined members
$$M[[ALLOCATE_ON_DEVICE(<GP_TYPE>, mcon->d_<GP_NAME>, mcon->d_size);]]

#endif  // LAMBDA_SHARE_ONE_MESSAGE_ARRAY
}

void MessageContentManager::Deallocate(MessageContent *mcon) {
#ifdef LAMBDA_SHARE_ONE_MESSAGE_ARRAY

  DEALLOCATE_ON_DEVICE(mcon->d_space);

#else  // Not share one array

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  DEALLOCATE_ON_DEVICE(mcon->d_is_full);
#endif

  //// TODO(laigd): add user defined members
$$M[[DEALLOCATE_ON_DEVICE(mcon->d_<GP_NAME>);]]

#endif
}

void MessageContentManager::Shuffle(
    MessageContent *mcon,
    thrust::device_ptr<unsigned int> thr_shuffle_index,
    void *d_tmp_buf) {
#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  SHUFFLE_MEMBER(bool,         mcon->d_is_full, mcon->d_size, d_tmp_buf, thr_shuffle_index);
#endif

  //// TODO(laigd): add user defined members
$$M[[SHUFFLE_MEMBER(<GP_TYPE>, mcon->d_<GP_NAME>, mcon->d_size, d_tmp_buf, thr_shuffle_index);]]
}

void MessageContentManager::Copy(
    const MessageContent &from,
    MessageContent *to) {
#ifdef LAMBDA_SHARE_ONE_MESSAGE_ARRAY

  COPY_FROM_DEVICE_TO_DEVICE(from, to, d_space, 0, 0, from.d_space_size, unsigned int);

#else  // Not share one array

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  COPY_FROM_DEVICE_TO_DEVICE(from, to, d_is_full, 0, 0, from.d_size, bool        );
#endif

  //// TODO(laigd): add user defined members
$$M[[COPY_FROM_DEVICE_TO_DEVICE(from, to, d_<GP_NAME>, 0, 0, from.d_size, <GP_TYPE>);]]

#endif
}

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
void MessageContentManager::Clear(MessageContent *mcon) {
  thrust::device_ptr<bool> thr_is_full(mcon->d_is_full);
  thrust::fill(thr_is_full, thr_is_full + mcon->d_size, false);
}
#endif

#ifdef LAMBDA_DEBUG
void MessageContentManager::DebugOutput(
    const MessageContent &mcon, const bool is_send_buf) {
  unsigned int *buf = NULL;
  checkCudaErrors(hipHostMalloc(&buf, mcon.d_size * sizeof(unsigned int)));

  cout << LAMBDA_HEADER << "[MessageContent "
       << (is_send_buf ? "Send" : "Recv") << "]" << endl;
#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  DEBUG_OUTPUT(buf, mcon.d_is_full, "is_full: ", mcon.d_size, bool);
#endif

  //// TODO(laigd): add user defined members
$$M[[DEBUG_OUTPUT(buf, mcon.d_<GP_NAME>, "<GP_NAME>: ", mcon.d_size, <GP_TYPE>);]]

  checkCudaErrors(hipHostFree(buf));
}
#endif

#ifdef LAMBDA_DEBUG
#undef LAMBDA_HEADER
#endif

#undef COPY_FROM_DEVICE_TO_DEVICE
